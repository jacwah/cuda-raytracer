
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdint.h>
#include <stdio.h>
#include <time.h>
#include <errno.h>

#define BG_COLOR (float3{0.0f, 0.0f, 0.0f})

//#define SPHERE_CENTER (float3{0.0f, 0.0f, 1.0f})
#define SPHERE_CENTER (float3{0.0f, 0.0f, 1.0f})
#define SPHERE_RADIUS 1.0f
#define FG_COLOR (float3{0.0f, 0.0f, 1.0f})
#define DIFFUSE 1.0f
#define SPECULAR_C 1.0f
#define SPECULAR_K 50

#define LIGHT_POS (float3{5.0f, 5.0f, -10.0f})
#define LIGHT_COLOR (float3{1.0f, 1.0f, 1.0f})
#define AMBIENT_COLOR (float3{0.05f, 0.05f, 0.05f})

#define CAMERA_POS (float3{0.0f, 0.0f, -1.0f})
#define CAMERA_FOCUS (float3{0.0f, 0.0f, 0.0f})

#define NUM_STREAMS 2

__device__ float3 operator*(float a, float3 v)
{
    return {a*v.x, a*v.y, a*v.z};
}

__device__ float3 operator+(float3 v, float3 w)
{
    return {v.x+w.x, v.y+w.y, v.z+w.z};
}

__device__ float3 operator+=(float3& v, float3 w)
{
    v.x += w.x;
    v.y += w.y;
    v.z += w.z;
    return v;
}

__device__ float3 operator-(float3 v, float3 w)
{
    return {v.x-w.x, v.y-w.y, v.z-w.z};
}

__device__ float3 normalize(float3 v)
{
    float rn = rnorm3df(v.x, v.y, v.z);
    return rn*v;
}

__device__ float dot(float3 v, float3 w)
{
    return v.x*w.x + v.y*w.y + v.z*w.z;
}

__device__ float3 clamp(float3 v)
{
    return {__saturatef(v.x), __saturatef(v.y), __saturatef(v.z)};
}

__device__ float intersect_sphere(float3 origin, float3 dir, float3 center, float radius)
{
    float3 os = origin - center;
    float b = 2.0f * dot(dir, os);
    float c = dot(os, os) - radius*radius;
    float d = b*b - 4*c;
    if (d > 0.0f) {
        float ds = sqrtf(d);
        float q;
        if (b < 0.0f)
            q = (-b - ds) / 2.0f;
        else
            q = (-b + ds) / 2.0f;
        float t0 = q;
        float t1 = c / q;
        if (t0 > t1) {
            float t2 = t0;
            t0 = t1;
            t1 = t2;
        }
        if (t1 >= 0.0f) {
            if (t0 < 0.0f)
                return t1;
            else
                return t0;
        }
    }
    return INFINITY;
}

__device__ float3 trace_ray(float3 origin, float3 dir)
{
    float t = intersect_sphere(origin, dir, SPHERE_CENTER, SPHERE_RADIUS);
    if (isinf(t))
        return BG_COLOR;

    float3 intersect = origin + t*dir;
    float3 n = normalize(intersect - SPHERE_CENTER);
    float3 toL = normalize(LIGHT_POS - intersect);
    float3 toO = normalize(origin - intersect);

    float3 color = AMBIENT_COLOR;

    float diffusity = dot(n, toL);
    if (diffusity > 0.0f)
        color += DIFFUSE*diffusity*FG_COLOR;

    float specularity = dot(n, normalize(toL + toO));
    if (specularity > 0.0f)
        color += SPECULAR_C*powf(specularity, SPECULAR_K)*LIGHT_COLOR;

    return clamp(color);
}

__global__ void render_image(uint8_t *image, unsigned w, unsigned h, unsigned line_off)
{
    const unsigned i = blockDim.x*blockIdx.x + threadIdx.x;
    const unsigned j = blockDim.y*(blockIdx.y+line_off) + threadIdx.y;

    if (i >= w || j >= h)
        return;

    float x =  2.0f*(i - (w-1)/2.0f) / (w-1.0f);
    float y = -2.0f*(j - (h-1)/2.0f) / (h-1.0f);

    float3 focus = CAMERA_FOCUS;
    focus.x = x;
    focus.y = y;
    float3 dir = normalize(focus - CAMERA_POS);

    float3 color = trace_ray(CAMERA_POS, dir);

    size_t idx = j*w + i;
    image[3*idx+0] = 255.99f*color.x;
    image[3*idx+1] = 255.99f*color.y;
    image[3*idx+2] = 255.99f*color.z;
}

double get_time()
{
    timespec now;
    clock_gettime(CLOCK_MONOTONIC, &now);
    return now.tv_sec + 1e-9*now.tv_nsec;
}

int main(int argc, char **argv)
{
    if (argc != 5) {
        fprintf(stderr, "usage: %s w h fname block\n", argv[0]);
        return 1;
    }

    unsigned w = atoi(argv[1]);
    unsigned h = atoi(argv[2]);
    const char *fname = argv[3];
    unsigned block = atoi(argv[4]);

    if (block*block > 1024) {
        fprintf(stderr, "block too large\n");
        return 1;
    }

    double start = get_time();

    FILE *fp = fopen(fname, "w");
    if (!fp) {
        perror("error opening file");
        return 1;
    }

    fprintf(fp, "P6\n");
    fprintf(fp, "%lu %lu\n", w, h);
    fprintf(fp, "255\n", w, h);

    unsigned lines_per_chunk = 1;
    size_t chunk_size = 3UL*block*w*lines_per_chunk;

    fprintf(stderr, "chunk size: %lu\n", chunk_size);

    size_t image_size = 3UL*w*h;
    size_t written = 0;
    uint8_t *chunk_buf;
    hipHostMalloc(&chunk_buf, NUM_STREAMS*chunk_size, hipHostMallocDefault);

    uint8_t *image_d;
    hipMalloc(&image_d, image_size);

    dim3 block_size(block, block);
    dim3 grid((w+block-1)/block, lines_per_chunk);
    unsigned image_lines = (h+block-1)/block;

    hipStream_t streams[NUM_STREAMS];
    for (int i = 0; i < NUM_STREAMS; i++) {
        hipStreamCreate(&streams[i]);
        render_image<<<grid, block_size, 0, streams[i]>>>(image_d, w, h, i*lines_per_chunk);
        hipMemcpyAsync(&chunk_buf[i*chunk_size], &image_d[i*chunk_size], chunk_size, hipMemcpyDefault, streams[i]);
    }
    unsigned next_line = lines_per_chunk*NUM_STREAMS;
    unsigned next_stream = 0;

    double after_init = get_time();

    while (written < image_size) {
        hipStreamSynchronize(streams[next_stream]);

        size_t write_size = chunk_size;
        if (written + write_size > image_size)
            write_size = image_size - written;
        size_t write = fwrite(&chunk_buf[next_stream*chunk_size], 1, write_size, fp);
        written += write_size;

        if (write != write_size) {
            perror("fwrite");
            return 1;
        }

        if (next_line < image_lines) {
            render_image<<<grid, block_size, 0, streams[next_stream]>>>(image_d, w, h, next_line);
            hipMemcpyAsync(&chunk_buf[next_stream*chunk_size], &image_d[(next_line/lines_per_chunk)*chunk_size], chunk_size, hipMemcpyDefault, streams[next_stream]);
            next_line += lines_per_chunk;
        }

        next_stream = (next_stream+1) % NUM_STREAMS;
    }

    double after_exec = get_time();

    fclose(fp);

    double end = get_time();

    printf("Total time:  %e\n", end-start);
    printf("---------------------\n");
    printf("init:  %e\n", after_init-start);
    printf("exec:  %e\n", after_exec-after_init);
    printf("write: %e\n", end-after_exec);
}
